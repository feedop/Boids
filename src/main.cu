﻿
#include "hip/hip_runtime.h"
#include ""

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <stdio.h>
#include <iostream>

#define DEFAULT_BOIDCOUNT 2

void handleInput(int argc, char* argv[], int& boidCount);

void usage()
{
    std::cout << "USAGE: ./Boids [boidCount]\n";
    exit(0);
}

int main(int argc, char* argv[])
{
    int boidCount;
    hipError_t cudaStatus;

    handleInput(argc, argv, boidCount);
    
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return 1;
    }

    GLFWwindow* window;

    /* Initialize the library */
    if (!glfwInit())
        return -1;
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    /* Create a windowed mode window and its OpenGL context */
    window = glfwCreateWindow(800, 800, "Boids", NULL, NULL);
    if (!window)
    {
        glfwTerminate();
        return -1;
    }

    /* Make the window's context current */
    glfwMakeContextCurrent(window);

    // Load GL and set the viewport to match window size
    gladLoadGL();
    glViewport(0, 0, 800, 800);

    /* Loop until the user closes the window */
    while (!glfwWindowShouldClose(window))
    {
        /* Render here */
        glClearColor(0.08f, 0.17f, 0.43f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        /* Swap front and back buffers */
        glfwSwapBuffers(window);

        /* Poll for and process events */
        glfwPollEvents();
    }

    // cleanup
    glfwTerminate();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void handleInput(int argc, char* argv[], int& boidCount)
{
    if (argc == 2)
    {
        boidCount = atoi(argv[1]);
        if (boidCount <= 0)
        {
            usage();
        }
    }
    else if (argc > 2)
    {
        usage();
    }
    else
    {
        boidCount = DEFAULT_BOIDCOUNT;
    }
}